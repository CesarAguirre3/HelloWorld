#include "hip/hip_runtime.h"


#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}

int main(int argc, char **argv)
{
	//launch the Kernel
	hello<<<NUM_BLOCKS, BLOCK_WIDTH >>>();

	//force the printf()s to flush
	hipDeviceSynchronize();
	
	printf("That's all!\n");

	return(0);
}